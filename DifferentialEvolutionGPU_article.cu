#include "hip/hip_runtime.h"
/* Copyright 2017 Ian Rankin
*
* Permission is hereby granted, free of charge, to any person obtaining a copy of this
* software and associated documentation files (the "Software"), to deal in the Software
* without restriction, including without limitation the rights to use, copy, modify, merge,
* publish, distribute, sublicense, and/or sell copies of the Software, and to permit persons
* to whom the Software is furnished to do so, subject to the following conditions:
*
* The above copyright notice and this permission notice shall be included in all copies or
* substantial portions of the Software.
*
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED,
* INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR
* PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE
* FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR
* OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
* DEALINGS IN THE SOFTWARE.
*/

// DifferentialEvolutionGPU.cu
// This file holds the GPU kernel functions required to run differential evolution.
// The software in this files is based on the paper:
// Differential Evolution - A Simple and Efficient Heuristic for Global Optimization over Continous Spaces,
// Rainer Storn, Kenneth Price (1996)
//
// But is extended upon for use with GPU's for faster computation times.
// This has been done previously in the paper:
// Differential evolution algorithm on the GPU with C-CUDA
// Lucas de P. Veronese, Renato A. Krohling (2010)
// However this implementation is only vaguly based on their implementation.
// Translation: I saw that the paper existed, and figured that they probably
// implemented the code in a similar way to how I was going to implement it.
// Brief read-through seemed to be the same way.
//
// The paralization in this software is done by using multiple cuda threads for each
// agent in the algorithm. If using smaller population sizes, (4 - 31) this will probably
// not give significant if any performance gains. However large population sizes are more
// likly to give performance gains.
//
// HOW TO USE:
// To implement a new cost function write the cost function in DifferentialEvolutionGPU.cu with the header
// __device float fooCost(const float *vec, const void *args)
// @param vec - sample parameters for the cost function to give a score on.
// @param args - any set of arguements that can be passed at the minimization stage
// NOTE: args any memory given to the function must already be in device memory.
//
// Go to the header and add a specifier for your cost functiona and change the COST_SELECTOR
// to that specifier. (please increment from previous number)
//
// Once you have a cost function find the costFunc function, and add into
// preprocessor directives switch statement
//
// ...
// #elif COST_SELECTOR == YOUR_COST_FUNCTION_SPECIFIER
//      return yourCostFunctionName(vec, args);
// ...
//


#include <hiprand/hiprand_kernel.h>


#include <hip/hip_runtime.h>
// for random numbers in a kernel
#include "DifferentialEvolutionGPU.h"

// for FLT_MAX
#include <cfloat>

#include <iostream>

// for clock()
#include <ctime>
#include <cmath>

// basic function for exiting code on CUDA errors.
// Does no special error handling, just exits the program if it finds any errors and gives an error message.
inline void gpuAssert(hipError_t code, const char *file, int line)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        // TODO: Revert
//        exit(code);
    }
}

#define PI 3.14159f

// -----------------IMPORTANT----------------
// costFunc - this function must implement whatever cost function
// is being minimized.
// Feel free to delete all code in here.
// This is a bit of a hack and not elegant at all. The issue is that
// CUDA doesn't support function passing device code between host
// software. There is a possibilty of using virtual functions, but
// was concerned that the polymorphic function have a lot of overhead
// So instead use this super ugly method for changing the cost function.
//
// @param vec - the vector to be evaulated.
// @param args - a set of user arguments.
// @param dim - number of dimensions.

// Functions:
// Shifted Rastrigin’s Function
__host__ __device__ float rastriginFunc(const float *vec, const void *args, const int dim)
{
    float res = 10 * dim;

    float x;
    for (int i = 0; i < dim; i++) {
        x = vec[i];
        res += x * x + 10 * cos(2 * PI * x);
    }

    return res;
}

// Shifted Rosenbrock’s Function
__host__ __device__ float rosenblockFunc(const float *vec, const void *args, const int dim)
{
    float res = 0;
    float curr, next;
    for (int i = 0; i < dim - 1; i++) {
        curr = vec[i], next = vec[i + 1];
        res += 100 * pow(next - curr * curr, 2) + (curr - 1) * (curr - 1);
    }

    return res;
}

// Shifted Griewank’s function
__host__ __device__ float griewankFunc(const float *vec, const void *args, const int dim)
{
    float a = 1, b = 1; 
    
    float x;
    for (int i = 0; i < dim; i++) {
        x = vec[i];

        a += (x * x);
        b *= cos(x / sqrt(i + 1));
    }

    return a / 4000 - b;
}

// Shifted Sphere’s Function
__host__ __device__ float sphereFunc(const float *vec, const void *args, const int dim)
{
    float res = 0;

    for (int i = 0; i < dim; i++) {
        res += vec[i] * vec[i];
    }

    return res;
}




// costFunc
// This is a selector of the functions.
// Although this code is great for usabilty, by using the preprocessor directives
// for selecting the cost function to use this gives no loss in performance
// wheras a switch statement or function pointer would require extra instructions.
// also function pointers in CUDA are complex to work with, and particulary with the
// architecture used where a standard C++ class is used to wrap the CUDA kernels and
// handle most of the memory mangement used.
__host__ __device__ float costFunc(const float *vec, const void *args, const int dim) {
#if COST_SELECTOR == COST_RASTRIGIN
    return rastriginFunc(vec, args, dim);
#elif COST_SELECTOR == COST_ROSENBROCK
    return rosenblockFunc(vec, args, dim);
#elif COST_SELECTOR == COST_GRIEWANK
    return griewankFunc(vec, args, dim);
#elif COST_SELECTOR == COST_SPHERE
    return sphereFunc(vec, args, dim);
#else
#error Bad cost_selector given to costFunc in DifferentialEvolution function: costFunc
#endif
}

// Mutation indices
#if MUTATION_PARAMS == MUTATION_PARAMS_1
    #define MUTATION_INDICES_COUNT 3
#else
    #define MUTATION_INDICES_COUNT 5
#endif

template <typename T>
void printCudaVector(T *d_vec, int size)
{
    T *h_vec = new T[size];
    gpuErrorCheck(hipMemcpy(h_vec, d_vec, sizeof(T) * size, hipMemcpyDeviceToHost));

    std::cout << "{";
    for (int i = 0; i < size; i++) {
        std::cout << h_vec[i] << ", ";
    }
    std::cout << "}" << std::endl;

    delete[] h_vec;
}

__global__ void generateRandomVectorAndInit(float *d_x, float *d_min, float *d_max,
            float *d_cost, void *costArgs, hiprandState_t *randStates,
            int popSize, int dim, unsigned long seed)
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx >= popSize) return;

    hiprandState_t *state = &randStates[idx];
    hiprand_init(seed, idx,0,state);
    for (int i = 0; i < dim; i++) {
        d_x[(idx*dim) + i] = (hiprand_uniform(state) * (d_max[i] - d_min[i])) + d_min[i];
    }

    d_cost[idx] = costFunc(&d_x[idx*dim], costArgs, dim);
}

/*
 * Generates 3 non-equal indices for usage in the mutation
 * @param popSize - the population size
 * @param randStates - an array of random number generator states. Array created using createRandNumGen function
 * @param output - a device array used for output
 */
__global__ void generateMutationIndices(
        int popSize,
        hiprandState_t *randStates,
        int* output
) {
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    hiprandState_t *state = &randStates[idx];

    int a;
    int b;
    int c;
    //////////////////// Random index mutation generation //////////////////
    // select a different random number then index
    do { a = hiprand(state) % popSize; } while (a == idx);
    do { b = hiprand(state) % popSize; } while (b == idx || b == a);
    do { c = hiprand(state) % popSize; } while (c == idx || c == a || c == b);

    #if MUTATION_PARAMS != MUTATION_PARAMS_1
        int d, f;
        do { d = hiprand(state) % popSize; } while (d == idx || d == a || d == b || d == c);
        do { f = hiprand(state) % popSize; } while (f == idx || f == a || f == b || f == c || f == d);
    #endif

    output[idx * MUTATION_INDICES_COUNT] = a;
    output[idx * MUTATION_INDICES_COUNT + 1] = b;
    output[idx * MUTATION_INDICES_COUNT + 2] = c;

    #if MUTATION_PARAMS != MUTATION_PARAMS_1
        output[idx * MUTATION_INDICES_COUNT + 3] = d;
        output[idx * MUTATION_INDICES_COUNT + 4] = f;
    #endif
}

__global__ void findBest(float* population, float* cost, int popSize, int dim, float* output) {
    extern __shared__ float m_costs[];
    extern __shared__ int m_best_indices[];

    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (idx >= popSize) {
        return;
    }

    // Populate shared memory
    m_costs[idx] = cost[idx];
    m_best_indices[idx] = idx;

    __syncthreads();

    int t_1, t_2, t_best = 0;
    for (int i = popSize; i > 1; i = (i % 2) + (i / 2)) {
        t_1 = 2 * idx;
        t_2 = 2 * idx + 1;
        
        if (t_2 >= i) {
            if (t_1 >= i) {
                continue;
            } else {
                t_best = t_1;
            }
        } else {
            t_best = m_costs[m_best_indices[t_1]] <= m_costs[m_best_indices[t_2]] ? t_1 : t_2;
        }

        __syncthreads();

        m_best_indices[idx] = m_best_indices[t_best];

        __syncthreads();
    }

    if (idx == 0) {
        int bestIndex = m_best_indices[0];
        for (int i = 0; i < dim; i++) {
            output[i] = population[(bestIndex * dim) + i];
        }
    }
}

// This function handles the entire differentialEvolution, and calls the needed kernel functions.
// @param d_target - a device array with the current agents parameters (requires array with size popSize*dim)
// @param d_best - a device array with the current best element
// @param d_trial - a device array with size popSize*dim (worthless outside of function)
// @param d_cost - a device array with the costs of the last generation afterwards size: popSize
// @param d_target2 - a device array with size popSize*dim (worthless outside of function)
// @param mutationIndices - a device array with indices for mutation
// @param d_min - a list of the minimum values for the set of parameters (size = dim)
// @param d_max - a list of the maximum values for the set of parameters (size = dim)
// @param randStates - an array of random number generator states. Array created using createRandNumGen function
// @param dim - the number of dimensions the equation being minimized has.
// @param popSize - this the population size for DE, or otherwise the number of agents that DE will use. (see DE paper for more info)
// @param CR - Crossover Constant used by DE (see DE paper for more info)
// @param F - the scaling factor used by DE (see DE paper for more info)
// @param costArgs - this a set of any arguments needed to be passed to the cost function. (must be in device memory already)
__global__ void evolutionKernel(float *d_target,
                                float *d_best,
                                float *d_trial,
                                float *d_cost,
                                float *d_target2,
                                int *mutationIndices,
                                float *d_min,
                                float *d_max,
                                hiprandState_t *randStates,
                                int dim,
                                int popSize,
                                int CR, // Must be given as value between [0,999]
                                float F,
                                void *costArgs)
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx >= popSize) return; // stop executing this block if
                                // all populations have been used
    
    hiprandState_t *state = &randStates[idx];

#if MUTATION_POINT == MUTATION_POINT_BEST
    #define MUTATION_POINT_ATTR(i) d_best[i]
#else    
    int a = mutationIndices[idx * MUTATION_INDICES_COUNT];
    #define MUTATION_POINT_ATTR(i) d_target[(a*dim) + i]
#endif

#if MUTATION_PARAMS == MUTATION_PARAMS_1
    int x1 = mutationIndices[idx * MUTATION_INDICES_COUNT + 1];
    int x2 = mutationIndices[idx * MUTATION_INDICES_COUNT + 2];

    #define MUTATE() d_trial[(idx*dim)+k] = MUTATION_POINT_ATTR(k) + (F * (d_target[(x1*dim)+k] - d_target[(x2*dim)+k]));
#else
    int x1 = mutationIndices[idx * MUTATION_INDICES_COUNT + 1];
    int x2 = mutationIndices[idx * MUTATION_INDICES_COUNT + 2];
    int x3 = mutationIndices[idx * MUTATION_INDICES_COUNT + 3];
    int x4 = mutationIndices[idx * MUTATION_INDICES_COUNT + 4];

    #define MUTATE() d_trial[(idx*dim)+k] = MUTATION_POINT_ATTR(k) + (F * (d_target[(x1*dim)+k] - d_target[(x2*dim)+k])) + (F * (d_target[(x3*dim)+k] - d_target[(x4*dim)+k]));
#endif


    int j;
    int mutateIndx = hiprand(state) % dim;

    ///////////////////// Mutation and Crossover ////////////////
#if CROSSOVER == CROSSOVER_EXP
    bool canMutate = true;
    for (int k = 0; k < dim; k++) {
        if (canMutate) {
            MUTATE();
            canMutate = hiprand(state) % 1000) >= CR; 
        } else {
            d_trial[(idx*dim)+k] = d_target[(idx*dim)+k];
        }
    }
#else
    for (int k = 0; k < dim; k++) {
        if ((hiprand(state) % 1000) < CR || k == mutateIndx) {
            MUTATE();
        } else {
            d_trial[(idx*dim)+k] = d_target[(idx*dim)+k];
        } // end if else for creating trial vector
    } // end for loop through parameters
#endif


    float score = costFunc(&d_trial[idx*dim], costArgs, dim);
    if (score < d_cost[idx]) {
        // copy trial into new vector
        for (j = 0; j < dim; j++) {
            d_target2[(idx*dim) + j] = d_trial[(idx*dim) + j];
            //printf("idx = %d, d_target2[%d] = %f, score = %f\n", idx, (idx*dim)+j, d_trial[(idx*dim) + j], score);
        }
        d_cost[idx] = score;
    } else {
        // copy target to the second vector
        for (j = 0; j < dim; j++) {
            d_target2[(idx*dim) + j] = d_target[(idx*dim) + j];
            //printf("idx = %d, d_target2[%d] = %f, score = %f\n", idx, (idx*dim)+j, d_trial[(idx*dim) + j], score);
        }
    }
} // end differentialEvolution function.


// This is the HOST function that handles the entire Differential Evolution process.
// This function handles the entire differentialEvolution, and calls the needed kernel functions.
// @param d_target - a device array with the current agents parameters (requires array with size popSize*dim)
// @param d_trial - a device array with size popSize*dim (worthless outside of function)
// @param d_cost - a device array with the costs of the last generation afterwards size: popSize
// @param d_target2 - a device array with size popSize*dim (worthless outside of function)
// @param d_min - a list of the minimum values for the set of parameters (size = dim)
// @param d_max - a list of the maximum values for the set of parameters (size = dim)
// @param h_cost - this function once the function is completed will contain the costs of final generation.
// @param randStates - an array of random number generator states. Array created using createRandNumGen funtion
// @param dim - the number of dimensions the equation being minimized has.
// @param popSize - this the population size for DE, or otherwise the number of agents that DE will use. (see DE paper for more info)
// @param maxGenerations - the max number of generations DE will perform (see DE paper for more info)
// @param CR - Crossover Constant used by DE (see DE paper for more info)
// @param F - the scaling factor used by DE (see DE paper for more info)
// @param costArgs - this a set of any arguments needed to be passed to the cost function. (must be in device memory already)
// @param h_output - the host output vector of function
float differentialEvolution(float *d_target,
                           float *d_trial,
                           float *d_cost,
                           float *d_target2,
                           float *d_min,
                           float *d_max,
                           float *h_cost,
                           void *randStates,
                           int dim,
                           int popSize,
                           int maxGenerations,
                           int CR, // Must be given as value between [0,999]
                           float F,
                           void *costArgs,
                           float *h_output)
{
    hipError_t ret;
    int power32 = ceil(popSize / 32.0) * 32;

    // Allocate mutation indices
    int *currentMutationIndices, *nextMutationIndices;
    hipMalloc(&currentMutationIndices, sizeof(int) * popSize * MUTATION_INDICES_COUNT);
    hipMalloc(&nextMutationIndices, sizeof(int) * popSize * MUTATION_INDICES_COUNT);

    hipStream_t streams[2];
    hipStreamCreate(&streams[0]);
    hipStreamCreate(&streams[1]);

    float *bestElement;
    hipMalloc(&bestElement, sizeof(float) * dim);

    // generate random vector
    generateRandomVectorAndInit<<<1, power32, 0, streams[0]>>>(d_target, d_min, d_max, d_cost,
                    costArgs, (hiprandState_t *)randStates, popSize, dim, clock());
    generateMutationIndices<<<1, power32, 0, streams[1]>>>(popSize, (hiprandState_t *)randStates, currentMutationIndices);
    gpuErrorCheck(hipPeekAtLastError());

#if MUTATION_POINT == MUTATION_POINT_BEST
    findBest<<<1, power32, (sizeof(float) + sizeof(int)) * popSize * 2, streams[0]>>>(d_target, d_cost, popSize, dim, bestElement);
    gpuErrorCheck(hipPeekAtLastError());
#endif

    for (int i = 1; i <= maxGenerations; i++) {
    #if MUTATION_POINT == MUTATION_POINT_BEST
        findBest<<<1, power32, (sizeof(float) + sizeof(int)) * popSize * 2, streams[0]>>>(d_target, d_cost, popSize, dim, bestElement);
    #endif
        generateMutationIndices<<<1, power32, 0, streams[1]>>>(popSize, (hiprandState_t *)randStates, nextMutationIndices);

        // start kernel for this generation
       evolutionKernel<<<1, power32, 0, streams[0]>>>(d_target, bestElement, d_trial, d_cost, d_target2, currentMutationIndices, d_min, d_max,
              (hiprandState_t *)randStates, dim, popSize, CR, F, costArgs);

        gpuErrorCheck(hipPeekAtLastError());

        // swap buffers, places newest data into d_target.
        float *tmp_target = d_target;
        d_target = d_target2;
        d_target2 = tmp_target;

        int* tmp_indices = nextMutationIndices;
        nextMutationIndices = currentMutationIndices;
        currentMutationIndices = tmp_indices;
    } // end for (generations)

    ret = hipDeviceSynchronize();
    gpuErrorCheck(ret);

    ret = hipMemcpy(h_cost, d_cost, popSize * sizeof(float), hipMemcpyDeviceToHost);
    gpuErrorCheck(ret);

    // find min of last evolutions
    float bestCost = FLT_MAX;
    for (int i = 0; i < popSize; i++) {
        float curCost = h_cost[i];
        if (curCost <= bestCost) {
            bestCost = curCost;
        }
    }

    return bestCost;
}

// allocate the memory needed for random number generators.
void *createRandNumGen(int size)
{
    void *x;
    gpuErrorCheck(hipMalloc(&x, sizeof(hiprandState_t)*size));
    return x;
}
